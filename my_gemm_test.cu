#include "hip/hip_runtime.h"
#include <iomanip>
#include <iostream>
#include <hiprand.h>
#include <hipblas.h>
#include <chrono>

#include "cute_impl.cuh"

#define CUBLAS_CHECK(call) { \
    hipblasStatus_t err; \
    if ((err = (call)) != HIPBLAS_STATUS_SUCCESS) { \
        std::cerr << "CUBLAS error in file '" << __FILE__ \
                  << "' at line " << __LINE__ << ": " \
                  << err << std::endl; \
        exit(1); \
    } \
}

class Timer {
    std::chrono::time_point<std::chrono::high_resolution_clock> start;
    public:
    Timer() : start(std::chrono::high_resolution_clock::now()) {}
    void reset() {
        start = std::chrono::high_resolution_clock::now();
    }
    double get() {
        auto end = std::chrono::high_resolution_clock::now();
        return std::chrono::duration<double>(end - start).count();
    }
};

template <typename T>
void run_gemms(int m, int n, int k, int reps) {
  T *A, *B, *C, *C_ref;
  hipMalloc(&A, m * k * sizeof(T));
  hipMalloc(&B, k * n * sizeof(T));
  hipMalloc(&C, m * n * sizeof(T));
  hipMalloc(&C_ref, m * n * sizeof(T));

  double n_flops = 2.0 * m * n * k;

  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, 42);
  if constexpr (std::is_same_v<T, float>) {
    static_assert(std::is_same_v<T, float>);
    hiprandGenerateUniform(gen, A, m * k);
    hiprandGenerateUniform(gen, B, k * n);
  } else if constexpr (std::is_same_v<T, double>) {
    hiprandGenerateUniformDouble(gen, A, m * k);
    hiprandGenerateUniformDouble(gen, B, k * n);
  }

  hipMemset(C, 0, m * n * sizeof(T));

  T alpha = 1.0;
  T beta = 0.0;
  {
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  CUBLAS_CHECK(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k, &alpha, A, m, B, n, &beta, C_ref, m));
  
  hipDeviceSynchronize();
  Timer timer;
  for (int i = 0; i < reps; i++) {
    CUBLAS_CHECK(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k, &alpha, A, m, B, n, &beta, C, m));
  }
  hipDeviceSynchronize();
  double per_gemm = timer.get() / reps;
  std::cout << "Time per CUBLAS GEMM: " << per_gemm << " s, GFLOPS: " << n_flops / per_gemm / 1e9 << std::endl;
  }

  {
    hipMemset(C, 0, m * n * sizeof(T));
    gemm('N', 'T', m, n, k, alpha, A, m, B, k, beta, C, m);
    hipDeviceSynchronize();
    Timer timer;
    for (int i = 0; i < reps; i++) {
        gemm('N', 'T', m, n, k, alpha, A, m, B, k, beta, C, m);
    }
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error in file '" << __FILE__ << "' at line " << __LINE__ << ": " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
    double per_gemm = timer.get() / reps;
    std::cout << "Time per CuTE GEMM: " << per_gemm
              << " s, GFLOPS: " << n_flops / per_gemm / 1e9 << std::endl;

    // Check result
    std::vector<T> C_host(m * n, 0);
    std::vector<T> C_ref_host(m * n, 0);
    hipMemcpy(C_host.data(), C, m * n * sizeof(T), hipMemcpyDeviceToHost);
    hipMemcpy(C_ref_host.data(), C_ref, m * n * sizeof(T), hipMemcpyDeviceToHost);
    double max_diff = 0;
    double max_val = 0;
    for (int i = 0; i < m * n; i++) {
      max_diff = std::max(max_diff, std::abs(C_host[i] - C_ref_host[i]));
      max_val = std::max(max_val, std::abs(C_ref_host[i]));
    }
    std::cout << "Max diff: " << std::setprecision(15) << max_diff << ", Max val: " << max_val << std::endl;
  }

  {
    hipMemset(C, 0, m * n * sizeof(T));
    my_gemm('N', 'T', m, n, k, alpha, A, m, B, k, beta, C, m);
    hipDeviceSynchronize();
    Timer timer;
    for (int i = 0; i < reps; i++) {
        my_gemm('N', 'T', m, n, k, alpha, A, m, B, k, beta, C, m);
    }
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error in file '" << __FILE__ << "' at line " << __LINE__ << ": " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
    double per_gemm = timer.get() / reps;
    std::cout << "Time per MyCuTE GEMM: " << per_gemm
              << " s, GFLOPS: " << n_flops / per_gemm / 1e9 << std::endl;

    // Check result
    std::vector<T> C_host(m * n, 0);
    std::vector<T> C_ref_host(m * n, 0);
    hipMemcpy(C_host.data(), C, m * n * sizeof(T), hipMemcpyDeviceToHost);
    hipMemcpy(C_ref_host.data(), C_ref, m * n * sizeof(T), hipMemcpyDeviceToHost);
    double max_diff = 0;
    double max_val = 0;
    for (int i = 0; i < m * n; i++) {
      max_diff = std::max(max_diff, std::abs(C_host[i] - C_ref_host[i]));
      max_val = std::max(max_val, std::abs(C_ref_host[i]));
    }
    std::cout << "Max diff: " << std::setprecision(15) << max_diff << ", Max val: " << max_val << std::endl;
  }
}

int main(int argc, char *argv[])
{
  int m = 512;
  int n = 512;
  int k = 512;
  int reps = 10;

  if (argc >= 2)
    m = atoi(argv[1]);
  if (argc >= 3)
    n = atoi(argv[2]);
  if (argc >= 4)
    k = atoi(argv[3]);

  std::cout << "m = " << m << ", n = " << n << ", k = " << k << ", reps = " << reps << std::endl;

  using T = double;
  run_gemms<T>(m, n, k, reps);
}
